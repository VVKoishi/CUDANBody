
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>
#include <iostream>
#include <chrono>

#define SOFTENING 1e-9f

/*
* Each body contains x, y, and z coordinate positions,
* as well as velocities in the x, y, and z directions.
*/
typedef struct { float x, y, z, vx, vy, vz; } Body;

/*
* Do not modify this function. A constraint of this exercise is
* that it remain a host function.
*/

void randomizeBodies(float *data, int n) {
	for (int i = 0; i < n; i++) {
		data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
	}
}

/*
* This function calculates the gravitational impact of all bodies in the system
* on all others, but does not update their positions.
*/

void bodyForce(Body *p, float dt, int n) {
	for (int i = 0; i < n; ++i) {
		float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

		for (int j = 0; j < n; j++) {
			float dx = p[j].x - p[i].x;
			float dy = p[j].y - p[i].y;
			float dz = p[j].z - p[i].z;
			float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
			float invDist = rsqrtf(distSqr);
			float invDist3 = invDist * invDist * invDist;

			Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
		}

		p[i].vx += dt * Fx; p[i].vy += dt * Fy; p[i].vz += dt * Fz;
	}
}

int main(const int argc, const char** argv) {

	/*
	 * Do not change the value for `nBodies` here. If you would like to modify it,
	 * pass values into the command line.
	 */

	int nBodies = 2 << 11;
	int salt = 0;
	if (argc > 1) nBodies = 2 << atoi(argv[1]);

	/*
	 * This salt is for assessment reasons. Tampering with it will result in automatic failure.
	 */

	if (argc > 2) salt = atoi(argv[2]);

	const float dt = 0.01f; // time step
	const int nIters = 50;  // simulation iterations

	int bytes = nBodies * sizeof(Body);
	float *buf;

	buf = (float *)malloc(bytes);

	Body *p = (Body*)buf;

	/*
	 * As a constraint of this exercise, `randomizeBodies` must remain a host function.
	 */

	randomizeBodies(buf, 6 * nBodies); // Init pos / vel data

	double totalTime = 0.0;

	auto beginTime = std::chrono::high_resolution_clock::now();

	/*
	 * This simulation will run for 10 cycles of time, calculating gravitational
	 * interaction amongst bodies, and adjusting their positions to reflect.
	 */

	/*******************************************************************/
	// Do not modify this line of code.
	for (int iter = 0; iter < nIters; iter++) {
		/*******************************************************************/

		/*
		* You will likely wish to refactor the work being done in `bodyForce`,
		* as well as the work to integrate the positions.
		*/

		bodyForce(p, dt, nBodies); // compute interbody forces

		/*
		* This position integration cannot occur until this round of `bodyForce` has completed.
		* Also, the next round of `bodyForce` cannot begin until the integration is complete.
		*/

		for (int i = 0; i < nBodies; i++) { // integrate position
			p[i].x += p[i].vx*dt;
			p[i].y += p[i].vy*dt;
			p[i].z += p[i].vz*dt;
		}
	}

	auto endTime = std::chrono::high_resolution_clock::now();
	std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(endTime - beginTime).count() << "msec\n";

	free(buf);
}