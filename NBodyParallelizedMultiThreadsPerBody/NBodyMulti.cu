#include "hip/hip_runtime.h"
// shared memory + loop optimization + multi threads per body

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>
#include <iostream>
#include <algorithm>
#include <chrono>
#include "hip/hip_runtime.h"
#include ""


#define SOFTENING 1e-9f

/*
* Each body contains x, y, and z coordinate positions,
* as well as velocities in the x, y, and z directions.
*/
typedef struct { float x, y, z, vx, vy, vz; } Body;

/*
* Do not modify this function. A constraint of this exercise is
* that it remain a host function.
*/

void randomizeBodies(float *data, int n) {
	for (int i = 0; i < n; i++) {
		data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
	}
}

/*
* This function calculates the gravitational impact of all bodies in the system
* on all others, but does not update their positions.
*/

__device__ float3
bodyBodyInteraction(float3 bi, float3 bj, float3 ai)
{
	float3 r;

	// r_ij [3 FLOPS]
	r.x = bj.x - bi.x;
	r.y = bj.y - bi.y;
	r.z = bj.z - bi.z;

	// distSqr = dot(r_ij, r_ij) + EPS^2 [6 FLOPS]
	float distSqr = r.x * r.x + r.y * r.y + r.z * r.z + SOFTENING;

	// invDistCube =1/distSqr^(3/2) [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
	float distSixth = distSqr * distSqr * distSqr;
	float invDistCube = rsqrtf(distSixth);

	// m = 1 
	// s = m_j * invDistCube [1 FLOP]
	// float s = bj.w * invDistCube;

	// a_i = a_i + s * r_ij [6 FLOPS]
	ai.x += r.x * invDistCube;
	ai.y += r.y * invDistCube;
	ai.z += r.z * invDistCube;
	return ai;
}

__device__ float3
tile_calculation(float3 myPosition, float3 accel, int numThreadsPerBody)
{
	int i;
	int p = blockDim.x / numThreadsPerBody;
	int starti = (threadIdx.x / p) * (p / numThreadsPerBody);
	extern __shared__ float3 shPosition[];	// shared memory

	#pragma unroll 32						// loop optimization
	for (i = starti; i < starti + (p / numThreadsPerBody); i++) {
		accel = bodyBodyInteraction(myPosition, shPosition[i], accel);
	}
	return accel;
}

__global__ void
calculate_forces(Body* ptr, float dt, int N, int numThreadsPerBody)
{
	extern __shared__ float3 shPosition[]; // p body shared memory pointer

	int p = blockDim.x / numThreadsPerBody;	// p rows/bodys = num bodys per block
	// int gtid = blockIdx.x * blockDim.x + threadIdx.x; // this thread index
	int gbid = blockIdx.x * p + threadIdx.x % p; // this body index
	float3 myPosition = { ptr[gbid].x, ptr[gbid].y, ptr[gbid].z }; // each body self pos
	float3 acc = { 0.0f, 0.0f, 0.0f }; // each body acc results

	for (int i = 0, int tile = 0; i < N; i += p, tile++) {
		int idx = tile * p + threadIdx.x;
		if (threadIdx.x < p) // divergence but desirable
			shPosition[threadIdx.x] = { ptr[idx].x, ptr[idx].y, ptr[idx].z };
		__syncthreads();
		acc = tile_calculation(myPosition, acc, numThreadsPerBody);
		__syncthreads();
	}
	// Save the result in global memory for the integration step.
	atomicAdd(&ptr[gbid].vx, acc.x * dt);
	atomicAdd(&ptr[gbid].vy, acc.y * dt);
	atomicAdd(&ptr[gbid].vz, acc.z * dt);
}

__global__ void integrate_position(Body *p, float dt, int n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n)
    {
        p[i].x += p[i].vx * dt;
        p[i].y += p[i].vy * dt;
        p[i].z += p[i].vz * dt;
    }
}

int main(const int argc, const char** argv) {

	/*
	 * Do not change the value for `nBodies` here. If you would like to modify it,
	 * pass values into the command line.
	 */

	int nBodies = 4096;
	int salt = 0;
	if (argc > 1) nBodies = 2 << atoi(argv[1]);

	/*
	 * This salt is for assessment reasons. Tampering with it will result in automatic failure.
	 */

	if (argc > 2) salt = atoi(argv[2]);

	const float dt = 0.01f; // time step
	const int nIters = 10;  // simulation iterations

	int bytes = nBodies * sizeof(Body);
	float *buf;

	buf = (float *)malloc(bytes);

	Body *p = (Body*)buf;

	/*
	 * As a constraint of this exercise, `randomizeBodies` must remain a host function.
	 */

	randomizeBodies(buf, 6 * nBodies); // Init pos / vel data

	double totalTime = 0.0;

	auto beginTime = std::chrono::high_resolution_clock::now();

	/*
	 * This simulation will run for 10 cycles of time, calculating gravitational
	 * interaction amongst bodies, and adjusting their positions to reflect.
	 */

	int numBodysPerBlock = 256;
	int numThreadsPerBody = 4;
	
	int blockSize = numBodysPerBlock * numThreadsPerBody;
	int numBlocks = (nBodies + numBodysPerBlock - 1) / numBodysPerBlock;
    int numTiles = numBlocks;
    int sharedMemSize = numBodysPerBlock * 3 * sizeof(float); // 3 floats for pos

	for (int iter = 0; iter < nIters; iter++) {
		calculate_forces <<< numBlocks, blockSize, sharedMemSize >>> (p, dt, nBodies, numThreadsPerBody);
		integrate_position <<< numBlocks, numBodysPerBlock >>> (p, dt, nBodies);
	}

	auto endTime = std::chrono::high_resolution_clock::now();
	auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - beginTime).count();
	std::cout << ms << "msec\n";

	double gflops = 1e-6 * nBodies * nBodies / ms * 19 * nIters;
	std::cout << gflops << "gflops\n";

	free(buf);
}